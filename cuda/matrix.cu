
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;
#define TILE_WIDTH 32

__global__ 
void MatrixMulKernel(float* M, float* N, float* P, int Width)
{
  __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
  __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x;  int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;

  int Row = by * blockDim.y + ty;
  int Col = bx * blockDim.x + tx;
  float Pvalue = 0;

  for (int p = 0; p < Width/TILE_WIDTH; ++p) {
    ds_M[ty][tx] = M[Row*Width + p*TILE_WIDTH+tx];
    ds_N[ty][tx] = N[(p*TILE_WIDTH+ty)*Width + Col];

    __syncthreads();
    for (int i = 0; i < TILE_WIDTH; ++i)
      Pvalue += ds_M[ty][i] * ds_N[i][tx];
    __syncthreads();
  }
  
  P[Row*Width+Col] = Pvalue;
}



int main()
{
  const int size = 1024;
  float* M = new float[size*size];
  float* N = new float[size*size];
  float* P = new float[size*size];

  hipMallocManaged(&M, size*size*sizeof(float));
  hipMallocManaged(&N, size*size*sizeof(float));
  hipMallocManaged(&P, size*size*sizeof(float));

  for (int i = 0; i < size*size; i++) {
    M[i] = 1.0;
    N[i] = 1.0;
  }

  dim3 DimGrid(ceil(size/TILE_WIDTH), ceil(size/TILE_WIDTH), 1);
  dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);

  MatrixMulKernel<<<DimGrid,DimBlock>>>(M,N,P,size);
  hipDeviceSynchronize();

  cout << P[0]<< endl;

  hipFree(M);
  hipFree(N);
  hipFree(P);
}
